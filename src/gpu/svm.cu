#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <mpi.h>

extern "C" {
  // #include <float/float32.h>
  // #include <float/slapack.h>
#include <Rinternals.h>
#include <stdlib.h>

#include "../common.h"
#include "../mpi_utils.h"
#include "../nm.h"
}

#include "blas.hh"
#include "cu_utils.hh"



typedef struct {
  hipblasHandle_t handle;
  int m;
  int n;
  const double *__restrict__ x;
  const int *__restrict__ y;
  double *__restrict__ w;
  double *__restrict__ work;
  double *__restrict__ s;
  MPI_Comm *__restrict__ comm;
} svm_param_t;




static inline double euc_norm_sq(hipblasHandle_t handle, const int n, const double *const __restrict__ x)
{
  double norm;
  hipblasStatus_t ret = hipblasDnrm2(handle, n, x, 1, &norm);
  
  return norm;
}



__global__ static void hinge_loss_sum(double *s, const int m, const int *const __restrict__ y, const double *const __restrict__ work)
{
  int tid = threadIdx.x;
  int i = tid + blockIdx.x*blockDim.x;
  
  if (i >= m)
    return;
  
  __shared__ double temp[TPB];
  
  double tmp = 1.0 - y[i]*work[i];
  if (tmp < 0.0)
    temp[tid] = 0.0;  
  else
    temp[tid] = tmp;
  
  __syncthreads();
  
  if (tid == 0)
  {
    double sum = 0.0;
    for (int i=0; i<TPB; i++)
      sum += temp[i];
    
    atomicAdd(s, sum);
  }
}



static inline double svm_cost(hipblasHandle_t handle,
  const int m, const int n, const double *const __restrict__ x,
  const int *const __restrict__ y, const double *const __restrict__ w,
  double *const __restrict__ s, double *const __restrict__ work,
  const MPI_Comm *const __restrict__ comm)
{
  int check;
  double J;
  double norm;
  double s_cpu = 0.0;
  
  hipMemset(s, 0, 1*sizeof(*s));
  
  // J_local = 1/m * sum(hinge_loss(1.0 - DATA(y)*matmult(DATA(x), w)))
  int nb = m / TPB;
  if (m % TPB)
    nb++;
  
  norm = euc_norm_sq(handle, n, w);
  
  mvm(handle, m, n, x, w, work);
  hinge_loss_sum<<<nb, TPB>>>(s, m, y, work);
  hipMemcpy(&s_cpu, s, sizeof(*s), hipMemcpyDeviceToHost);
  J = ((double) 1.0/m) * s_cpu;
  
  // J = allreduce(J_local) + 1/m * 0.5 * norm2(w)
  check = MPI_Allreduce(MPI_IN_PLACE, &J, 1, MPI_DOUBLE, MPI_SUM, *comm);
  MPI_CHECK(comm, check);
  
  J += ((double) 1.0/m) * 0.5 * norm;
  
  return J;
}



static inline void svm_nmwrap(int n, point_t *point, const void *arg)
{
  const svm_param_t *args = (const svm_param_t*) arg;
  hipMemcpy(args->w, point->x, n*sizeof(double), hipMemcpyHostToDevice);
  point->fx = svm_cost(args->handle, args->m, n, args->x, args->y, args->w, args->s, args->work, args->comm);
  hipMemcpy(point->x, args->w, n*sizeof(double), hipMemcpyDeviceToHost);
}



static inline void svm(const int m, const int n, const double *const __restrict__ x,
  const int *const __restrict__ y, double *const __restrict__ w, MPI_Comm *const __restrict__ comm,
  optimset_t *const __restrict__ optimset)
{
  svm_param_t args;
  point_t start, solution;
  
  
  hipblasHandle_t handle;
  hipblasStatus_t st = hipblasCreate(&handle);
  if (st != HIPBLAS_STATUS_SUCCESS)
    error("hipblasCreate() failed\n");
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
  
  double *x_gpu;
  int *y_gpu;
  double *w_gpu;
  double *work_gpu;
  double *s_gpu;
  
  hipMalloc(&x_gpu, m*n*sizeof(*x_gpu));
  hipMalloc(&y_gpu, m*sizeof(*y_gpu));
  hipMalloc(&w_gpu, n*sizeof(*w_gpu));
  hipMalloc(&work_gpu, m*sizeof(*work_gpu));
  hipMalloc(&s_gpu, sizeof(*s_gpu));
  
  if (x_gpu == NULL || y_gpu == NULL || w_gpu == NULL || work_gpu == NULL || s_gpu == NULL)
  {
    CUFREE(x_gpu);
    CUFREE(y_gpu);
    CUFREE(w_gpu);
    CUFREE(work_gpu);
    CUFREE(s_gpu);
    error("Unable to allocate device memory");
  }
  
  hipMemcpy(x_gpu, x, m*n*sizeof(*x), hipMemcpyHostToDevice);
  hipMemcpy(y_gpu, y, m*sizeof(*y), hipMemcpyHostToDevice);
  
  start.x = w;
  memset(w, 0, n*sizeof(*w));
  
  args.handle = handle;
  args.m = m;
  args.n = n;
  args.x = x_gpu;
  args.y = y_gpu;
  args.w = w_gpu;
  args.work = work_gpu;
  args.comm = comm;
  
  nelder_mead(n, &start, &solution, &svm_nmwrap, &args, optimset);
  
  for (int i=0; i<n; i++)
    w[i] = solution.x[i];
  
  hipblasDestroy(handle);
  
  hipFree(x_gpu);
  hipFree(y_gpu);
  hipFree(w_gpu);
  hipFree(work_gpu);
  
  free(solution.x);
}



extern "C" SEXP R_svm(SEXP x, SEXP y, SEXP maxiter, SEXP comm_)
{
  SEXP ret, ret_names, w, niters;
  optimset_t opts;
  MPI_Comm *comm = get_mpi_comm_from_Robj(comm_);
  const int m = nrows(x);
  const int n = ncols(x);
  
  PROTECT(ret = allocVector(VECSXP, 2));
  PROTECT(ret_names = allocVector(STRSXP, 2));
  PROTECT(w = allocVector(REALSXP, n));
  PROTECT(niters = allocVector(INTSXP, 1));
  
  SET_VECTOR_ELT(ret, 0, w);
  SET_VECTOR_ELT(ret, 1, niters);
  SET_STRING_ELT(ret_names, 0, mkChar("w"));
  SET_STRING_ELT(ret_names, 1, mkChar("niters"));
  setAttrib(ret, R_NamesSymbol, ret_names);
  
  set_nm_opts(INTEGER(maxiter)[0], &opts);
  svm(m, n, REAL(x), INTEGER(y), REAL(w), comm, &opts);
  
  UNPROTECT(4);
  return ret;
}
